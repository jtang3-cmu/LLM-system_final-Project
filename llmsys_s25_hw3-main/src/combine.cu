#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {
    /**
     * A naive (non-tiled) GPU kernel for matrix multiplication:
     * Each thread handles one element [row, col] in the out matrix
     *
     * a_shape = [batch_size, m, n]
     * b_shape = [batch_size, n, p]
     * out_shape = [batch_size, m, p]
     */
    // BEGIN ASSIGN1_2
    // (1) Identify batch, row, col
    int batch = blockIdx.z;                // which batch among batch_size
    int row   = blockDim.x * blockIdx.x + threadIdx.x;  // 0..m-1
    int col   = blockDim.y * blockIdx.y + threadIdx.y;  // 0..p-1

    // (2) Gather shapes for convenience
    int m = a_shape[1];
    int n = a_shape[2];  // same as b_shape[1]
    int p = b_shape[2];  // out_shape[2]

    // (3) If out-of-range, do nothing
    if (row >= m || col >= p) return;

    // (4) We'll do a sum across dimension n
    float val = 0.0f;
    for (int k = 0; k < n; k++) {
        // Build a 3D index for A
        int a_idx[3] = {batch, row, k};
        // Build a 3D index for B
        int b_idx[3] = {batch, k, col};

        // Convert those to positions
        int a_pos = index_to_position(a_idx, a_strides, 3);
        int b_pos = index_to_position(b_idx, b_strides, 3);

        val += a_storage[a_pos] * b_storage[b_pos];
    }

    // (5) Write to out
    int out_idx[3] = {batch, row, col};
    int out_pos    = index_to_position(out_idx, out_strides, 3);
    out[out_pos]   = val;
    // END ASSIGN1_2
}



__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];
    
    // BEGIN ASSIGN1_2
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= out_size) return;

    // 1. Convert 'i' -> out_index
    to_index(i, out_shape, out_index, shape_size);
    // 2. Use broadcast if needed
    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
    // 3. Convert out_index -> out_pos
    int out_pos = index_to_position(out_index, out_strides, shape_size);
    // 4. Convert in_index -> in_pos
    int in_pos  = index_to_position(in_index, in_strides, shape_size);
    // 5. Apply unary function
    float xval  = in_storage[in_pos];
    out[out_pos] = fn(fn_id, xval, 0.0f);
    // END ASSIGN1_2
}



__global__ void reduceKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
    // __shared__ double cache[BLOCK_DIM]; // optional shared mem approach
    int out_index[MAX_DIMS];

    // BEGIN ASSIGN1_2
    // Each thread computes one output cell
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= out_size) return;

    // 1. Convert i -> out_index
    to_index(i, out_shape, out_index, shape_size);

    // 2. We'll accumulate into 'acc'
    float acc = reduce_value;

    // The length of the dimension we're reducing
    int len_dim = a_shape[reduce_dim];

    // 3. For each index along reduce_dim, accumulate
    for (int s = 0; s < len_dim; s++) {
        out_index[reduce_dim] = s;
        int a_pos = index_to_position(out_index, a_strides, shape_size);
        acc = fn(fn_id, acc, a_storage[a_pos]);
    }

    // 4. Reconstruct out_pos
    //    Must re-build out_index to original since we overwrote reduce_dim above
    to_index(i, out_shape, out_index, shape_size);
    int out_pos = index_to_position(out_index, out_strides, shape_size);

    // 5. Write to out
    out[out_pos] = acc;
    // END ASSIGN1_2
}

__global__ void zipKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    // BEGIN ASSIGN1_2
    // Each thread corresponds to one element in 'out'
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= out_size) return;

    // 1. Convert i -> out_index
    to_index(i, out_shape, out_index, out_shape_size);

    // 2. out_index -> out_pos
    int out_pos = index_to_position(out_index, out_strides, out_shape_size);

    // 3. broadcast -> a_index
    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
    int a_pos = index_to_position(a_index, a_strides, a_shape_size);

    // 4. broadcast -> b_index
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
    int b_pos = index_to_position(b_index, b_strides, b_shape_size);

    // 5. Apply fn
    float va = a_storage[a_pos];
    float vb = b_storage[b_pos];
    out[out_pos] = fn(fn_id, va, vb);
    // END ASSIGN1_2
}

extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}